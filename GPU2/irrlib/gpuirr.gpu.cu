#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <cutil.h>
#include "cuda_pointer.h"
#include "gnutil.h"

#define _out_
#define PROFILE

typedef DblFloat twofl;
typedef Gvec3<twofl> dvec3;
typedef Gvec3<float> fvec3;

#ifdef PROFILE
#include <sys/time.h>
static double get_wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.e-6 * tv.tv_usec;
}
#else
static double get_wtime(){
	return 0.0;
}
#endif

struct Jparticle{
	dvec3 pos;  // 6
	fvec3 vel;  // 9
	fvec3 acc2; // 12
	fvec3 jrk6; // 15
	float mass; // 16
	twofl time; // 18
	int   addr; // 19
	int   pad;  // 20

	__host__ Jparticle(
			const double _pos [3],
			const double _vel [3],
			const double _acc2[3],
			const double _jrk6[3],
			const double _mass,
			const double _time,
			const int    _addr) :
		pos(_pos), vel(_vel), acc2(_acc2), jrk6(_jrk6), mass(_mass), time(_time), addr(_addr) {}

	__device__ Jparticle(const void *vsrc){
		const float4 *src = (float4 *)vsrc;
		_out_ float4 *dst = (float4 *)this;
		dst[0] = src[0];
		dst[1] = src[1];
		dst[2] = src[2];
		dst[3] = src[3];
		dst[4] = src[4];
	}
	__device__ void store(void *vdst) const{
		const float4 *src = (float4 *)this;
		_out_ float4 *dst = (float4 *)vdst;
		dst[0] = src[0];
		dst[1] = src[1];
		dst[2] = src[2];
		dst[3] = src[3];
		dst[4] = src[4];
	}
};

struct Predictor{
	dvec3 pos;   // 6
	fvec3 vel;   // 9
	float mass;  // 10
	twofl tlast; // 12, just for padding

	__device__ Predictor(const Jparticle &p, const twofl &time)
	{
		pos   = p.pos;
		vel   = p.vel;
		mass  = p.mass;
		// tlast = time;

		const float dt = time - p.time;
		pos += dt*(p.vel + dt*(p.acc2 + dt*(p.jrk6)));
		vel += (2.f*dt)*(p.acc2 + (1.5f*dt)*(p.jrk6));
	}

	__host__ void print(const int i, FILE *fp = stdout) const{
		double x = pos.x;
		double y = pos.y;
		double z = pos.z;
		double vx = vel.x;
		double vy = vel.y;
		double vz = vel.z;
		fprintf(fp, "%4d : %f %f %f, %f %f %f, %f\n",
				i, x, y, z, vx, vy, vz, mass);
		fflush(fp);
	}
};

struct Force{
	fvec3 acc; // 3
	fvec3 jrk; // 6

	__device__ Force() : acc(0.f), jrk(0.f) {}
	__device__ Force(const Predictor &ip, const Predictor &jp)
	{
		const fvec3 dr = jp.pos - ip.pos;
		const fvec3 dv = jp.vel - ip.vel;
		const float r2 = dr * dr;
		const float rv = dr * dv;

		const float rinv   = rsqrt(r2);
		const float rinv2  = rinv * rinv;
		const float alpha  = -3.f * rv * rinv2;
		const float mrinv3 = jp.mass * rinv * rinv2;

		acc = mrinv3 * dr;
		jrk = mrinv3 * (dv + alpha * dr);
	}

	__device__ void operator += (const Force &rhs){
		acc += rhs.acc;
		jrk += rhs.jrk;
	}
};

struct NBlist{
	enum{ NB_MAX = 511 };
	int nnb;
	int nb[NB_MAX]; // 2 kBi

	__host__ void print(const int i, FILE *fp = stdout) const{
		fprintf(fp, "%6d%6d :", i, nnb);
		for(int k=0; k<nnb; k++){
			fprintf(fp, " %d", nb[k]);
		}
		fprintf(fp, "\n");
		fflush(fp);
	}
};

__global__ void kernel_jp_flush(
		const int nj,
		const Jparticle jpsrc[],
		_out_ Jparticle jpdst[])
{
#if 0
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < nj){
		const Jparticle jp = jpsrc[tid];
		jpdst[jp.addr] = jp;
	}
#else
	// 5 threads   / particle
	// 6 particles / warp
	__shared__ float4 f4share[64];
	const int tid = threadIdx.x;
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;
	const int wbid = gid / 32;
	const int wtid = gid % 32;
	const int pid = wtid / 5;
	const int qid = wtid % 5;
	const int iaddr = 6 * wbid + pid;
	if((iaddr < nj) && (pid < 6)){
		float4 *src = (float4 *)(jpsrc + iaddr);
		f4share[tid] = src[qid];
		const int paddr = 32*(tid/32) + 5*pid;
		const Jparticle &jp = *(Jparticle *)(f4share + paddr);
		const int jaddr = jp.addr;
		float4 *dst = (float4 *)(jpdst + jaddr);
		dst[qid] = f4share[tid];
	}
#endif
}

__global__ void kernel_list_flush(
		const int4   task  [],
		const int    list  [],
		_out_ NBlist nblist[])
{
	const int  bid    = blockIdx.x;
	const int  tid    = threadIdx.x;
	const int4 mytask = task[bid];
	const int  addr   = mytask.x;
	const int  nnb    = mytask.y;
	const int  off    = mytask.z;
	const int *src    = list + off;
	_out_ int *dst    = nblist[addr].nb;

	if(tid == 0) nblist[addr].nnb = nnb; // store

	for(int k=0; k<nnb; k+=blockDim.x){
		if(k+tid < nnb) dst[k+tid] = src[k+tid] - 1;
	}
}

__global__ void kernel_predict(
		const int       js,
		const int       je,
		const twofl     ti,
		const Jparticle ptcl[],
		_out_ Predictor pred[])
{
#if 0
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(js<=tid && tid<je){
		pred[tid] = Predictor(ptcl[tid], ti);
	}
#else
	const int tid = threadIdx.x;
	const int off = blockDim.x * blockIdx.x;
	const int nth = blockDim.x;
	__shared__ float4 sbuf[128*5];
	Jparticle *sptcl = (Jparticle *)sbuf;
	Predictor *spred = (Predictor *)sbuf;
	{   // LOAD
		float4 *src = (float4 *)(ptcl + off);
		float4 *dst = (float4 *)(sptcl);
#pragma unroll
		for(int k=0; k<5; k++, src+=nth, dst+=nth){
			dst[tid] = src[tid];
		}
	}

	__syncthreads();
	Predictor pp(sptcl[tid], ti);
	__syncthreads();
	spred[tid] = pp;
	__syncthreads();

	{   // STORE
		float4 *src = (float4 *)(spred);
		float4 *dst = (float4 *)(pred + off);
#pragma unroll
		for(int k=0; k<3; k++, src+=nth, dst+=nth){
			dst[tid] = src[tid];
		}
	}
#endif
}

__global__ void kernel_gravity(
		const int       iaddr[],
		const NBlist    list [],
		const Predictor pred [],
		_out_ Force     force[])
{
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	const int nth = blockDim.x;
	const int i   = iaddr[bid];
	const int nnb = list[i].nnb;
	const int *nblist = list[i].nb;

	__shared__ Force fshare[64];

	const Predictor ip = pred[i];
	Force fo; // flushed by constructor
	
	for(int k=0; k<nnb; k+=nth){
		if(k+tid < nnb){
			const int j = nblist[k+tid];
			const Predictor jp = pred[j];
			fo += Force(ip, jp);
		}
	}

	fshare[tid] = fo;

	if(nnb > 32){
		__syncthreads();
		if(tid < 32) fshare[tid] += fshare[tid + 32];
	}

	if(nnb > 16){
		if(tid < 16) fshare[tid] += fshare[tid + 16];
	}
	if(nnb > 8){
		if(tid <  8) fshare[tid] += fshare[tid +  8];
	}
	if(tid <  4) fshare[tid] += fshare[tid +  4];
	if(tid <  2) fshare[tid] += fshare[tid +  2];
	if(tid <  1) fshare[tid] += fshare[tid +  1];

	if(tid == 0){
		force[bid] = fshare[0];
	}
}

static double time_jp, time_list, time_pred, time_grav, time_onep;

struct Jparticle_que{
	enum{
		MAX_JP_QUE = (1<<14), // 1MBi
		NTHREAD    = 64,
	};
	typedef cudaPointer<Jparticle> pointer;

	int      count;
	pointer  buf;
	pointer *dst;

	Jparticle_que() : count(0), buf(), dst(NULL) {}
	void initialize(pointer * const _dst){
		count = 0;
		buf.allocate(MAX_JP_QUE);
		dst = _dst;
	}
	void finalize(){
		buf.free();
		count = 0;
		dst   = NULL;
	}
	void flush(){
		if(count > 0){
			const double t0 = get_wtime();

			buf.htod(count);
#if 0
			const int nblock = 1 + (count-1)/NTHREAD;
#else       // 5 threads   / particle
			// 6 particles / warp
			const int nwarp  = 1 + (count-1)/6;
			const int nblock = 1 + (nwarp-1)/(NTHREAD/32); 
#endif

			kernel_jp_flush <<<nblock, NTHREAD>>>
				(count, buf, (*dst));
			CUDA_SAFE_THREAD_SYNC();

			count = 0;

			const double t1 = get_wtime();
			::time_jp += t1-t0;
		}
	}
	void push(
		const int addr,
		const double pos [3],
		const double vel [3],
		const double acc2[3],
		const double jrk6[3],
		const double mass,
		const double time)
	{
		buf[count++] = Jparticle(pos, vel, acc2, jrk6, mass, time, addr);
		if(count >= MAX_JP_QUE) flush();
	}
};

struct List_que{
	enum{
		MAX_LIST_QUE = 4096,
		MAX_LIST_BUF = (1<<20), // 4 MBi
		NTHREAD      = 64,
	};
	int                  count;
	int                  list_len;
	cudaPointer<int>     list_buf;
	cudaPointer<int4>    task_buf; // addr, nnb, off, pad
	cudaPointer<NBlist> *dst;

	List_que() : count(0), list_len(0), list_buf(), task_buf(), dst(NULL) {}
	void initialize(cudaPointer<NBlist> * const _dst){
		dst      = _dst;
		count    = 0;
		list_len = 0;
		list_buf.allocate(MAX_LIST_BUF);
		task_buf.allocate(MAX_LIST_QUE);
	}
	void finalize(){
		dst      = NULL;
		count    = 0;
		list_len = 0;
		list_buf.free();
		task_buf.free();
	}
	void push(
			const int addr,
			const int nnb,
			const int nblist[])
	{
		if(list_len + nnb > MAX_LIST_BUF) flush();
		task_buf[count++] = make_int4(addr, nnb, list_len, 0);
		int *dst = &list_buf[list_len];
		for(int k=0; k<nnb; k++){
			dst[k] = nblist[k];
		}
		list_len += nnb;
		if(count >= MAX_LIST_QUE) flush();
	}
	void flush(){
		if(count > 0){
			const double t0 = get_wtime();

			list_buf.htod(list_len);
			task_buf.htod(count);

			const int nblock = count;
			kernel_list_flush <<<nblock, NTHREAD>>>
				(task_buf, list_buf, (*dst));
			CUDA_SAFE_THREAD_SYNC();

			count    = 0;
			list_len = 0;

			const double t1 = get_wtime();
			::time_list += t1-t0;
		}
	}
};

enum{
	NFORCE_MAX = 1024,
};

static cudaPointer<Jparticle> ptcl;
static cudaPointer<Predictor> pred;
static cudaPointer<Force    > force;
static cudaPointer<NBlist   > list;
static Jparticle_que          jpque;
static List_que               listque;
static cudaPointer<int      > iaddr;

static void gpuirr_open(
		const int nmax,
		const int lmax)
{
	assert(lmax <= 1 + NBlist::NB_MAX);

	fprintf(stderr, "**************************** \n"); 
	fprintf(stderr, "Opening GPUIRR lib. GPU ver. \n"); 
	fprintf(stderr, " nmax = %d, lmax = %d\n", nmax, lmax);
	fprintf(stderr, "**************************** \n"); 

	ptcl .allocate(nmax + 128);
	pred .allocate(nmax + 128);
	force.allocate(NFORCE_MAX);
	list .allocate(nmax);
	jpque  .initialize(&ptcl);
	listque.initialize(&list);
	iaddr.allocate(NFORCE_MAX);

	time_jp = time_list = time_pred = time_grav = time_onep = 0.0;
	fprintf(stderr, "Opened GPUIRR lib. GPU ver. \n"); 
}

static void gpuirr_close(){
	fprintf(stderr, "**************************** \n"); 
	fprintf(stderr, "Closing GPUIRR lib. GPU ver. \n"); 
	fprintf(stderr, "time jp    : %f sec\n", time_jp);
	fprintf(stderr, "time list  : %f sec\n", time_list);
	fprintf(stderr, "time pred  : %f sec\n", time_pred);
	fprintf(stderr, "time grav  : %f sec\n", time_grav);
	fprintf(stderr, "time onep  : %f sec\n", time_onep);
	fprintf(stderr, "**************************** \n"); 

	ptcl .free();
	pred .free();
	force.free();
	list .free();
	jpque  .finalize();
	listque.finalize();
	iaddr.free();
}

static void gpuirr_set_jp(
		const int addr,
		const double pos [3],
		const double vel [3],
		const double acc2[3],
		const double jrk6[3],
		const double mass,
		const double time)
{
	jpque.push(addr, pos, vel, acc2, jrk6, mass, time);
}

static void gpuirr_set_list(
		const int addr,
		const int nnb,
		const int nblist[])
{
	assert(nnb <= NBlist::NB_MAX);
	listque.push(addr, nnb, nblist);
}

static void gpuirr_pred_all(
		const int    js,
		const int    je,
		const double ti){
	jpque.flush();

	const double t0 = get_wtime();

	const int nthread = 128;
	const int nblock  = 1 + (je-1)/nthread;
	kernel_predict <<<nblock, nthread>>>
		(js, je, twofl(ti), ptcl, pred);
	CUDA_SAFE_THREAD_SYNC();

	const double t1 = get_wtime();
	::time_pred += t1-t0;
}

static void gpuirr_firr_vec(
		const int nitot,
		const int addr[],
		_out_ double accout[][3],
		_out_ double jrkout[][3])
{
	listque.flush();

	const double t0 = get_wtime();

	for(int ii=0; ii<nitot; ii+=NFORCE_MAX){
		const int ni = nitot-ii < NFORCE_MAX ? nitot-ii : NFORCE_MAX;
		for(int i=0; i<ni; i++){
			iaddr[i] = addr[ii + i] - 1;
		}
		iaddr.htod(ni);

		kernel_gravity <<<ni, 64>>> (iaddr, list, pred, force);

		force.dtoh(ni);
		for(int i=0; i<ni; i++){
			force[i].acc.write(accout[ii + i]);
			force[i].jrk.write(jrkout[ii + i]);
		}
	}

	const double t1 = get_wtime();
	if(nitot == 1){
		::time_onep += t1-t0;
	}else{
		::time_grav += t1-t0;
	}
}


// FORTRAN interfface
extern "C"{
	void gpuirr_open_(int *nmax, int *lmax){
		gpuirr_open(*nmax, *lmax);
	}
	void gpuirr_close_(){
		gpuirr_close();
	}
	void gpuirr_set_jp_(
		int    *addr,
		double  pos [3],
		double  vel [3],
		double  acc2[3],
		double  jrk6[3],
		double *mass,
		double *time)
	{
		gpuirr_set_jp((*addr)-1, pos, vel, acc2, jrk6, *mass, *time);
	}
	void gpuirr_set_list_(
		int *addr,
		int *nblist)
	{
		gpuirr_set_list((*addr)-1, *nblist, nblist+1);
	}
	void gpuirr_pred_all_(
			int    *js,
			int    *je,
			double *ti)
	{
		gpuirr_pred_all((*js)-1, *(je),  *ti);
	}
#if 1
	void gpuirr_firr_(
			int    *addr,
			double  acc[3],
			double  jrk[3])
	{
		gpuirr_firr_vec(1, addr, (double (*)[3])acc, (double (*)[3])jrk);
	}
#endif
	void gpuirr_firr_vec_(
			int   *ni,
			int    addr[],
			double acc [][3],
			double jrk [][3])
	{
		gpuirr_firr_vec(*ni, addr, acc, jrk);
	}

	void DEBUG_list(const int count){
		listque.flush();
		list.dtoh(count);
		for(int i=0; i<count; i++){
			list[i].print(i);
		}
	}
	void DEBUG_pred(const int count){
		pred.dtoh(count);
		for(int i=0; i<count; i++){
			pred[i].print(i);
		}
	}
}
