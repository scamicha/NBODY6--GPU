#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>

#include <cutil.h>

#include "cuda_pointer.h"
#include "gnutil.h"
#include "gninit.h"

#define PROFILE
#ifdef PROFILE
#include <sys/time.h>
static double get_wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.e-6 * tv.tv_usec;
}
struct Timer{
	const char *name;
	FILE       *fp;
	const char *format;
	double tstart;

	Timer(
			const char *_name,
			FILE       *_fp     = stdout,
			const char *_format = " %-10s : %f sec\n")
	   : name(_name), fp(_fp), format(_format)
	{
		tstart = get_wtime();
	}
	~Timer(){
		double tend = get_wtime();
		fprintf(fp, format, name, tend - tstart);
		fflush(fp);
	}
};
#else
static double get_wtime(){
	return 0.0;
}
struct Timer{
	Timer(
			const char *_name,
			FILE       *_fp     = stdout,
			const char *_fotmat = " %s : %f sec\n")
	{}
	~Timer(){}
};
#endif

typedef Gvec3<float   > fvec3;
typedef Gvec3<DblFloat> dvec3;
typedef DblFloat        twofl;

struct PosH2{
	dvec3 pos;
	float h2;
	float pad;

	__host__ PosH2(const double _pos[3], const double _h2) :
		pos(_pos), h2(_h2) {}
};

__global__ void kernel_count_neib(
		const int    nbody,
		const float  eps2,
		const PosH2  ptcl[],
		_out_ int    nnb_out[])
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid >= nbody) return;
	const PosH2 ip = ptcl[tid];
	int nnb = 0;

#pragma unroll 4
	for(int j=0; j<nbody; j++){
		const PosH2 jp = ptcl[j];
		const fvec3 dr = jp.pos - ip.pos;
		const float r2 = eps2 + dr*dr;
		if((j != tid) && (r2 < ip.h2)) nnb++;
	}

	nnb_out[tid] = nnb;
}

__global__ void kernel_get_neib(
		const int    nbody,
		const float   eps2,
		const PosH2  ptcl   [],
		const int    nboff  [],
		_out_ int    nnb_out[],
		_out_ int    nblist [])
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid >= nbody) return;
	const PosH2 ip = ptcl[tid];
	const int ioff = nboff[tid];
	int *nbdst = nblist + ioff;
	int nnb = 0;

#pragma unroll 4
	for(int j=0; j<nbody; j++){
		const PosH2 jp = ptcl[j];
		const fvec3 dr = jp.pos - ip.pos;
		const float r2 = eps2 + dr*dr;
		if((j != tid) && (r2 < ip.h2)){
			*nbdst++ = j;
			nnb++;
		}
	}

	nnb_out[tid] = nnb;
}

struct PosM{
	dvec3 pos;
	float mass;
	float pad;

	PosM(const double x[3], const double m) : pos(x), mass(m) {}
	PosM(int) : pos(dvec3(0.0)), mass(0.0f), pad(0.0f) {}
	__device__ PosM() {}
};

__global__ void kernel_calc_pot(
		const int    nbody,
		const float  eps2,
		const PosM   ptcl[],
		_out_ twofl  pot_out[])
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid >= nbody) return;
	const PosM ip = ptcl[tid];
	twofl pot(0.f, 0.f);

#pragma unroll 4
	for(int j=0; j<nbody; j++){
		const PosM  jp = ptcl[j];
		const fvec3 dr = jp.pos - ip.pos;
		const float r2 = eps2 + dr*dr;
		const float pij = jp.mass * rsqrtf(r2);
		if(j != tid) pot += pij;
	}

	pot_out[tid] = pot;
}

struct Particle{
	dvec3  pos;  // 6
	fvec3  vel;  // 9
	float  mass; // 10
	float2 pad;  // 12

	Particle(
			const double _pos[3], 
			const double _vel[3], 
			const double _mass) 
		: pos(_pos), vel(_vel), mass(_mass) {}
	__device__ Particle() {}
};

#if 0
struct Force{
	dvec3    acc; // 6
	fvec3    jrk; // 9
	float3   pad; // 12

	__device__ 
	Force() : acc(twofl(0.0f, 0.0f)), jrk(0.0f) {}

	__device__ 
	void accumulate(const fvec3 &a, const fvec3 &j)
	{
		acc += a;
		jrk += j;
	}
};
#else
struct Force{
	dvec3    acc; // 6
	dvec3    jrk; // 12

	__device__ 
	Force() : acc(twofl(0.0f, 0.0f)), jrk(twofl(0.0f, 0.0f)) {}

	__device__ 
	void accumulate(const fvec3 &a, const fvec3 &j)
	{
		acc += a;
		jrk += j;
	}
};
#endif

__global__ void kernel_calc_force(
		const int      nbody,
		const float    eps2,
		const Particle ptcl [],
		_out_ Force    force[])
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid >= nbody) return;
	const Particle ip = ptcl[tid];
	Force fo; // initialized by the constructor

#pragma unroll 4
	for(int j=0; j<nbody; j++){
		const Particle jp = ptcl[j];
		const fvec3 dr = jp.pos - ip.pos;
		const fvec3 dv = jp.vel - ip.vel;

		const float r2 = eps2 + dr * dr;
		const float rv = dr * dv;
		const float rinv1 = (tid != j) ? rsqrtf(r2) : 0.0f;

		const float rinv2 = rinv1 * rinv1;
		const float mrinv1 = jp.mass * rinv1;
		const float mrinv3 = mrinv1 * rinv2;
		const float alpha  = -3.f * rv * rinv2;

		const fvec3 acc = mrinv3 * dr;
		const fvec3 jrk = mrinv3 * (dv + alpha * dr);

		fo.accumulate(acc, jrk);
	}

	force[tid] = fo;
}

struct FatParticle{
	dvec3  pos;  // 6
	fvec3  vel;  // 9
	fvec3  acc;  // 12
	fvec3  jrk;  // 15
	float  h2;   // 16
	float  mass; // 17
	float3 pad;  // 20

	FatParticle(
			const double _pos[3], 
			const double _vel[3], 
			const double _acc[3], 
			const double _jrk[3], 
			const double _h2, 
			const double _mass) 
		: pos(_pos), vel(_vel), acc(_acc), jrk(_jrk), h2(_h2), mass(_mass) {}
	__device__ FatParticle() {}
};

struct FatForce{
	dvec3    acc; // 6
	fvec3    jrk; // 9
	fvec3    snp; // 12
	fvec3    crk; // 15
	float    pad; // 16

	__device__ 
	FatForce() : acc(twofl(0.0f, 0.0f)), jrk(0.0f), snp(0.0f), crk(0.0f) {}

	__device__ 
	void accumulate(
			const fvec3 &a, 
			const fvec3 &j, 
			const fvec3 &s, 
			const fvec3 &c)
	{
		acc += a;
		jrk += j;
		snp += s;
		crk += c;
	}
};

__global__ void kernel_calc_fpoly(
		const int         nbody,
		const float       eps2,
		const FatParticle ptcl[],
		_out_ FatForce    force_reg[],
		_out_ FatForce    force_irr[])
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid >= nbody) return;
	const FatParticle ip = ptcl[tid];
	FatForce freg; // initialized by the constructor
	FatForce firr; // initialized by the constructor

#pragma unroll 4
	for(int j=0; j<nbody; j++){
		const FatParticle jp = ptcl[j];
		const fvec3 dr = jp.pos - ip.pos;
		const fvec3 dv = jp.vel - ip.vel;
		const fvec3 da = jp.acc - ip.acc;
		const fvec3 dj = jp.jrk - ip.jrk;

		const float r2 = eps2 + dr * dr;
		const float rv = dr * dv;
		const float v2 = dv * dv;
		const float ra = dr * da;
		const float va = dv * da;
		const float rj = dr * dj;

		const float rinv1 = (tid != j) ? rsqrtf(r2) : 0.0f;

		const float rinv2 = rinv1 * rinv1;
		const float mrinv1 = jp.mass * rinv1;
		const float mrinv3 = mrinv1 * rinv2;

		const float alpha  = rv * rinv2;
		const float alphalpha = alpha * alpha;
		const float beta   = ((v2 + ra) * rinv2 + alphalpha);
		const float gamma  = ((3.f*va + rj) * rinv2 + alpha * (3.f*beta - 4.f*alphalpha));

		const fvec3 acc = mrinv3*dr;
		const fvec3 jrk = mrinv3*dv - (3.f*alpha)*acc;
		const fvec3 snp = mrinv3*da - (6.f*alpha)*jrk - (3.f*beta)*acc;
		const fvec3 crk = mrinv3*dj - (9.f*alpha)*snp - (9.f*beta)*jrk - (3.f*gamma)*acc;

		if(r2 < ip.h2){
			firr.accumulate(acc, jrk, snp, crk);
		}else{
			freg.accumulate(acc, jrk, snp, crk);
		}
	}

	force_reg[tid] = freg;
	force_irr[tid] = firr;
}

struct PosVelH2{
	dvec3 pos;
	fvec3 vel;
	float h2;
	float pad[2];

	PosVelH2(
			const double _pos[3], 
			const double _vel[3], 
			const double _h2) 
		: pos(_pos), vel(_vel), h2(_h2) {}
};

__global__ void kernel_calc_dtmin(
		const int         nbody,
		const float       eps2,
		const PosVelH2    ptcl[],
		_out_ float2      dtmin[])
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid >= nbody) return;
	const PosVelH2 ip = ptcl[tid];
	float dtreg = 16777216.;
	float dtirr = 16777216.;

#pragma unroll 4
	for(int j=0; j<nbody; j++){
		const PosVelH2 jp = ptcl[j];
		const fvec3 dr = jp.pos - ip.pos;
		const fvec3 dv = jp.vel - ip.vel;
		const float r2 = eps2 + dr * dr;
		const float v2 = dv * dv;
		const float dtij = r2 / v2;

		if(j != tid){
			if(r2 < ip.h2){
				dtirr = min(dtirr, dtij);
			}else{
				dtreg = min(dtreg, dtij);
			}
		}
	}
	dtmin[tid] = make_float2(sqrtf(dtreg), sqrtf(dtirr));
}

__global__ void kernel_count_friend(
		const int      nbody,
		const float    eps2,
		const PosVelH2 ptcl[],
		const float    dt_ov_eta,
		_out_ int      nfr_out[])
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid >= nbody) return;
	const PosVelH2 ip = ptcl[tid];
	const float crit = dt_ov_eta * dt_ov_eta;
	int nfr = 0;

#pragma unroll 4
	for(int j=0; j<nbody; j++){
		const PosVelH2 jp = ptcl[j];
		const fvec3 dr = jp.pos - ip.pos;
		const fvec3 dv = jp.vel - ip.vel;
		const float r2 = eps2 + dr * dr;
		const float v2 = dv * dv;
		const float dtij = r2 / v2;
		if((j != tid) && (dtij < crit)) nfr++;
	}

	nfr_out[tid] = nfr;
}

namespace gninit{
	void count_neib(
			const int    nbody,
			const double eps2,
			const double pos[][3],
			const double h2 [],
			_out_ int    nnb_out[])
	{
		Timer timer(__func__);

		cudaPointer<PosH2, true> posh2(nbody);
		cudaPointer<int  , true> nnb  (nbody);

		for(int i=0; i<nbody; i++){
			posh2[i] = PosH2(pos[i], h2[i]);
		}
		posh2.htod();

		const int nthread = 128;
		const int nblock  = 1 + (nbody-1)/nthread;
		kernel_count_neib <<<nblock, nthread>>>
			(nbody, eps2, posh2, nnb);

		nnb.dtoh();
		for(int i=0; i<nbody; i++){
			nnb_out[i] = nnb[i];
		}
	}

	void get_neib(
			const int    nbody,
			const double eps2,
			const double pos[][3],
			const double h2 [],
			const int    nnb_in[],
			_out_ int * const list[]) // array of pointers
	{
		Timer timer(__func__);

		cudaPointer<PosH2, true> posh2 (nbody);
		cudaPointer<int  , true> nboff (nbody);
		cudaPointer<int  , true> nnb   (nbody);
		cudaPointer<int  , true> nblist;

		int nbsum = 0;
		for(int i=0; i<nbody; i++){
			posh2[i] = PosH2(pos[i], h2[i]);
			nboff[i] = nbsum;
			nbsum += nnb_in[i];
		}
		nblist.allocate(nbsum);
		fprintf(stderr, "nbsum = %d\n", nbsum);

		posh2.htod();
		nboff.htod();

		const int nthread = 128;
		const int nblock  = 1 + (nbody-1)/nthread;
		kernel_get_neib <<<nblock, nthread>>>
			(nbody, eps2, posh2, nboff, nnb, nblist);

		nnb.dtoh();
		nblist.dtoh();

		for(int i=0; i<nbody; i++){
			assert(nnb_in[i] == nnb[i]);
			const int len = nnb[i];
			int *src = &nblist[nboff[i]];
			int *dst = list[i];
			for(int k=0; k<len; k++){
				dst[k] = src[k];
			}
		}
	}

	void calc_pot(
			const int    nbody,
			const double eps2,
			const double mass[],
			const double pos [][3],
			_out_ double pot_out[])
	{
		Timer timer(__func__);

		cudaPointer<PosM,  true> posm(nbody);
		cudaPointer<twofl, true> pot (nbody);

		for(int i=0; i<nbody; i++){
			posm[i] = PosM(pos[i], mass[i]);
		}
		posm.htod();

		const int nthread = 128;
		const int nblock  = 1 + (nbody-1)/nthread;
		kernel_calc_pot <<<nblock, nthread>>>
			(nbody, eps2, posm, pot);

		pot.dtoh();
		for(int i=0; i<nbody; i++){
			pot_out[i] = -double(pot[i]);
		}
	}

	void calc_force(
			const int    nbody,
			const double eps2,
			const double mass[],
			const double pos [][3],
			const double vel [][3],
			_out_ double acc [][3],
			_out_ double jrk [][3])
	{
		Timer timer(__func__);

		cudaPointer<Particle, true> ptcl (nbody);
		cudaPointer<Force,    true> force(nbody);

		for(int i=0; i<nbody; i++){
			ptcl[i] = Particle(pos[i], vel[i], mass[i]);
		}
		ptcl.htod();

		const int nthread = 128;
		const int nblock  = 1 + (nbody-1)/nthread;
		kernel_calc_force <<<nblock, nthread>>>
			(nbody, eps2, ptcl, force);

		force.dtoh();
		for(int i=0; i<nbody; i++){
			force[i].acc.write(acc[i]);
			force[i].jrk.write(jrk[i]);
		}
	}

	void calc_fpoly(
			const int    nbody,
			const double eps2,
			const double mass[],
			const double pos [][3],
			const double vel [][3],
			const double acc [][3],
			const double jrk [][3],
			const double h2 [],
			_out_ double fpoly_reg[][4][3],
			_out_ double fpoly_irr[][4][3])
	{
		Timer timer(__func__);

		cudaPointer<FatParticle, true> ptcl(nbody);
		cudaPointer<FatForce,    true> freg(nbody);
		cudaPointer<FatForce,    true> firr(nbody);

		for(int i=0; i<nbody; i++){
			ptcl[i] = FatParticle(pos[i], vel[i], 
					      acc[i], jrk[i], h2[i], mass[i]);
		}
		ptcl.htod();

		const int nthread = 64;
		const int nblock  = 1 + (nbody-1)/nthread;
		kernel_calc_fpoly <<<nblock, nthread>>>
			(nbody, eps2, ptcl, freg, firr);

		freg.dtoh();
		firr.dtoh();
		for(int i=0; i<nbody; i++){
			freg[i].acc.write(fpoly_reg[i][0]);
			freg[i].jrk.write(fpoly_reg[i][1]);
			freg[i].snp.write(fpoly_reg[i][2]);
			freg[i].crk.write(fpoly_reg[i][3]);
			firr[i].acc.write(fpoly_irr[i][0]);
			firr[i].jrk.write(fpoly_irr[i][1]);
			firr[i].snp.write(fpoly_irr[i][2]);
			firr[i].crk.write(fpoly_irr[i][3]);
		}
	}
	void calc_dtmin(
			const int    nbody,
			const double eps2,
			const double pos [][3],
			const double vel [][3],
			const double h2 [],
			_out_ double dtreg[],
			_out_ double dtirr[])
	{
		Timer timer(__func__);

		cudaPointer<PosVelH2, true> ptcl (nbody);
		cudaPointer<float2,   true> dtmin(nbody);

		for(int i=0; i<nbody; i++){
			ptcl[i] = PosVelH2(pos[i], vel[i], h2[i]);
		}
		ptcl.htod();

		const int nthread = 128;
		const int nblock  = 1 + (nbody-1)/nthread;
		kernel_calc_dtmin <<<nblock, nthread>>>
			(nbody, eps2, ptcl, dtmin);

		dtmin.dtoh();
		for(int i=0; i<nbody; i++){
			dtreg[i] = dtmin[i].x;
			dtirr[i] = dtmin[i].y;
		}
	}
	void count_friend(
			const int    nbody,
			const double eps2,
			const double pos [][3],
			const double vel [][3],
			const double dt_ov_eta,
			_out_ int    nfr_out[])
	{
		Timer timer(__func__);

		cudaPointer<PosVelH2, true> ptcl(nbody);
		cudaPointer<int   ,   true> nfr (nbody);

		for(int i=0; i<nbody; i++){
			ptcl[i] = PosVelH2(pos[i], vel[i], 0.0);
		}
		ptcl.htod();

		const int nthread = 128;
		const int nblock  = 1 + (nbody-1)/nthread;
		kernel_count_friend <<<nblock, nthread>>>
			(nbody, eps2, ptcl, dt_ov_eta, nfr);

		nfr.dtoh();
		for(int i=0; i<nbody; i++){
			nfr_out[i] = nfr[i];
		}
	}
}
