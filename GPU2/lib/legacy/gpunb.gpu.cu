#include "hip/hip_runtime.h"
// #include <iostream>
#include <stdio.h>
// #include <vector>
#include <cmath>
#include <cassert>
#include <cutil.h>
#include <omp.h>
#include "cuda_pointer.h"

#define NTHREAD 64 // 64, 96, 128 or 192
#define NJBLOCK 16 // 8800GTS/512 has 16
#define NIBLOCK 16 // 16 or 32 
#define NIMAX (NTHREAD * NIBLOCK) // 1024

#define NBMAX 64 // NNB per block

template <class T>
struct myvector{
	int num;
	T *val;
	myvector(){
		num = 0;
		val = NULL;
	}
	~myvector(){
		delete [] val;
	}
	void clear(){
		num = 0;
	}
	void reserve(size_t count){
		val = new T[count];
	}
	void free(){
		delete [] val;
	}
	void push_back(const T &t){
		val[num++] = t;
	}
	size_t size(){
		return num;
	}
	T &operator[](int i){
		return val[i];
	}
};

#define PROFILE
#ifdef PROFILE
#include <sys/time.h>
static double get_wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.e-6 * tv.tv_usec;
}
#else
static double get_wtime(){
	return 0.0;
}
#endif

static double time_send, time_grav;
static long long numInter;

struct Jparticle{
	float3 pos;
	float  mass;
	float3 vel;
	float  pad;
	Jparticle() {}
	Jparticle(double mj, double xj[3], double vj[3]){
		pos.x = xj[0];
		pos.y = xj[1];
		pos.z = xj[2];
		mass  = mj;
		vel.x = vj[0];
		vel.y = vj[1];
		vel.z = vj[2];
	}
};
struct Iparticle{
	float3 pos;
	float  h2;
	float3 vel;
	float  pad;
	Iparticle() {}
	Iparticle(double h2i, double xi[3], double vi[3]){
		pos.x = xi[0];
		pos.y = xi[1];
		pos.z = xi[2];
		h2    = h2i;
		vel.x = vi[0];
		vel.y = vi[1];
		vel.z = vi[2];
	}
};
struct Force{
	float3 acc;
	float  pot;
	float3 jrk;
	int    nnb;          //  8 words
	unsigned short  neib[NBMAX]; // 24 words
	__device__  Force(){
		acc.x = acc.y = acc.z = 0.f;
		jrk.x = jrk.y = jrk.z = 0.f;
		pot = 0.f;
		nnb = 0;
	}
};

__device__ float rsqrtfNR(float x){
	float y = rsqrtf(x);
	return (-0.5f * y) * (x*y*y - 3.0f);
}

#if 0
struct force1{
	float dx, dy, dz;
	float dvx, dvy, dvz;
	float r2;
	float rv;
	// __device__ force1(){}
	__device__ void calc(
			const Iparticle &ip,
			const Jparticle &jp){
		dx = jp.pos.x - ip.pos.x;
		dy = jp.pos.y - ip.pos.y;
		dz = jp.pos.z - ip.pos.z;
		dvx = jp.vel.x - ip.vel.x;
		dvy = jp.vel.y - ip.vel.y;
		dvz = jp.vel.z - ip.vel.z;
		r2 = dx*dx + dy*dy + dz*dz;
		rv = dx*dvx + dy*dvy + dz*dvz;
	}
};
struct force2{
	float rinv1;
	// __device__ force2(){}
	__device__ void calc(
			const int j,
			const Iparticle &ip,
			const force1 &f1,
			Force &fo){
		rinv1 = rsqrtf(f1.r2);
		if(f1.r2 < ip.h2){
			fo.neib[fo.nnb++ % NBMAX] = j;
			rinv1 = 0.f;
		}
	}
};
struct force3{
	float rinv1, rinv2, rinv3;
	float rv;
	// __device__ force3(){}
	__device__ void calc(
			const Jparticle &jp,
			const force1 &f1,
			const force2 &f2,
			Force &fo){
		rinv1 = f2.rinv1;
		rinv2 = rinv1 * rinv1;
		rinv1 *= jp.mass;
		rinv3 = rinv1 * rinv2;
		rv = f1.rv * -3.f * rinv2;

		fo.pot += rinv1;
		fo.acc.x += rinv3 * f1.dx;
		fo.acc.y += rinv3 * f1.dy;
		fo.acc.z += rinv3 * f1.dz;
		fo.jrk.x += rinv3 * (f1.dvx + rv * f1.dx);
		fo.jrk.y += rinv3 * (f1.dvy + rv * f1.dy);
		fo.jrk.z += rinv3 * (f1.dvz + rv * f1.dz);
	}
};
#endif

__device__ void h4_kernel(
		const int j,
		const Iparticle &ip, 
		const Jparticle &jp, 
		Force &fo){
	float dx = jp.pos.x - ip.pos.x;
	float dy = jp.pos.y - ip.pos.y;
	float dz = jp.pos.z - ip.pos.z;
	float dvx = jp.vel.x - ip.vel.x;
	float dvy = jp.vel.y - ip.vel.y;
	float dvz = jp.vel.z - ip.vel.z;

	float r2 = dx*dx + dy*dy + dz*dz;
	float rv = dx*dvx + dy*dvy + dz*dvz;
	float rinv1 = rsqrtf(r2);
	if(r2 < ip.h2){
		// fo.neib[fo.nnb++ % NBMAX] = j;
		fo.neib[fo.nnb & (NBMAX-1)] = (unsigned)j;
		fo.nnb++;
		rinv1 = 0.f;
	}
	float rinv2 = rinv1 * rinv1;
	float mrinv1 = jp.mass * rinv1;
	float mrinv3 = mrinv1 * rinv2;
	rv *= -3.f * rinv2;
	
#ifdef POTENTIAL
	fo.pot += mrinv1;
#endif
	fo.acc.x += mrinv3 * dx;
	fo.acc.y += mrinv3 * dy;
	fo.acc.z += mrinv3 * dz;
	// fo.acc.z += 1.0;
	fo.jrk.x += mrinv3 * (dvx + rv * dx);
	fo.jrk.y += mrinv3 * (dvy + rv * dy);
	fo.jrk.z += mrinv3 * (dvz + rv * dz);
}
__global__ void h4_gravity(
		int nbody,
		Iparticle ipbuf[],
		Jparticle jpbuf[],
		Force fobuf[][NJBLOCK]){
	int ibid = blockIdx.x;
	int jbid = blockIdx.y;
	int tid = threadIdx.x;
	int iaddr = tid + NTHREAD * ibid;
	int jstart = (nbody * (jbid  )) / NJBLOCK;
	int jend   = (nbody * (jbid+1)) / NJBLOCK;

	Iparticle ip = ipbuf[iaddr];
	Force fo;
	for(int j=jstart; j<jend; j+=NTHREAD){
		__shared__ Jparticle jpshare[NTHREAD];
		__syncthreads();
#if 0
		jpshare[tid] = jpbuf[j+tid];
#else
		float4 *src = (float4 *)&jpbuf[j];
		float4 *dst = (float4 *)jpshare;
		dst[        tid] = src[        tid];
		dst[NTHREAD+tid] = src[NTHREAD+tid];
#endif
		__syncthreads();

		if(jend-j < NTHREAD){
			for(int jj=0; jj<jend-j; jj++){
				Jparticle jp = jpshare[jj];
				h4_kernel(j+jj, ip, jp, fo);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<NTHREAD; jj++){
				Jparticle jp = jpshare[jj];
				h4_kernel(j+jj, ip, jp, fo);
			}
		}
	}
	fobuf[iaddr][jbid] = fo;
}

#if 0
static Jparticle *jp_host, *jp_dev;
static Iparticle *ip_host, *ip_dev;
static Force (*fo_host)[NJBLOCK], (*fo_dev)[NJBLOCK]; 
#else
static cudaPointer <Jparticle> jpbuf;
static cudaPointer <Iparticle> ipbuf;
static cudaPointer <Force[NJBLOCK]> fobuf;
#endif
#define MAX_CPU 8
static myvector<int> nblist[MAX_CPU];
static int nbody, nbodymax;
// static int *nblist;

void GPUNB_open(int nbmax){
	time_send = time_grav = 0.0;
	numInter = 0;
    // CUT_DEVICE_INIT();
	// size_t jpsize = nbmax * sizeof(Jparticle);
	// size_t ipsize = NIMAX * sizeof(Iparticle);
	// size_t fosize = NIBLOCK * NJBLOCK * NTHREAD * sizeof(Force);
	// hipHostMalloc((void **)&jp_host, jpsize);
	// jpsize += NTHREAD * sizeof(Jparticle);
	// hipMalloc    ((void **)&jp_dev , jpsize);
	// hipHostMalloc((void **)&ip_host, ipsize);
	// hipMalloc    ((void **)&ip_dev , ipsize);
	// hipHostMalloc((void **)&fo_host, fosize);
	// hipMalloc    ((void **)&fo_dev , fosize);
	jpbuf.allocate(nbmax + NTHREAD);
	ipbuf.allocate(NIMAX);
	fobuf.allocate(NIMAX);
	nbodymax = nbmax;
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		nblist[tid].reserve(nbmax);
	}
}
void GPUNB_close(){
	// hipHostFree(jp_host);
	// hipFree    (jp_dev);
	// hipHostFree(ip_host);
	// hipFree    (ip_dev);
	// hipHostFree(fo_host);
	// hipFree    (fo_dev);
	jpbuf.free();
	ipbuf.free();
	fobuf.free();
	nbodymax = 0;

#ifdef PROFILE
#if 0
	std::cerr << "***********************" << std::endl;
	std::cerr << "time send : " << time_send << " sec " << std::endl;
	std::cerr << "time grav : " << time_grav << " sec " << std::endl;
	std::cerr << 60.e-9 * numInter / time_grav << " Gflops (gravity part only)" << std::endl;
	std::cerr << "***********************" << std::endl;
#else
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "time send : %f sec\n", time_send);
	fprintf(stderr, "time grav : %f sec\n", time_grav);
	fprintf(stderr, "%f Gflops (gravity part only)\n", 60.e-9 * numInter / time_grav);
	fprintf(stderr, "***********************\n");
#endif
#endif
}
void GPUNB_send(
		int nj,
		double mj[],
		double xj[][3],
		double vj[][3]){
	time_send -= get_wtime();
	nbody = nj;
	assert(nbody <= nbodymax);
	for(int j=0; j<nj; j++){
		// jp_host[j] = Jparticle(mj[j], xj[j], vj[j]);
		jpbuf[j] = Jparticle(mj[j], xj[j], vj[j]);
	}
	// size_t jpsize = nj * sizeof(Jparticle);
	// hipMemcpy(jp_dev, jp_host, jpsize, hipMemcpyHostToDevice);
	jpbuf.htod(nj);
	time_send += get_wtime();
}
void GPUNB_regf(
		int ni,
		double h2[],
		double xi[][3],
		double vi[][3],
		double acc[][3],
		double jrk[][3],
		double pot[],
		int lmax,
		int nbmax,
		int *listbase){
	time_grav -= get_wtime();
	numInter += ni * nbody;
	assert(0 < ni && ni <= NIMAX);
	for(int i=0; i<ni; i++){
		// ip_host[i] = Iparticle(h2[i], xi[i], vi[i]);
		ipbuf[i] = Iparticle(h2[i], xi[i], vi[i]);
	}
	// set i-particles
	// size_t ipsize = ni * sizeof(Iparticle);
	// hipMemcpy(ip_dev, ip_host, ipsize, hipMemcpyHostToDevice);
	ipbuf.htod(ni);

	// gravity kernel
	int niblock = 1 + (ni-1) / NTHREAD;
	dim3 grid(niblock, NJBLOCK, 1);
	dim3 threads(NTHREAD, 1, 1);
#if 0
	int sharedMemSize = NTHREAD * sizeof(Jparticle);
	h4_gravity <<< grid, threads, sharedMemSize >>> 
		(nbody, ip_dev, jp_dev, fo_dev);
#else
	// h4_gravity <<< grid, threads >>> 
	//	(nbody, ip_dev, jp_dev, fo_dev);
	h4_gravity <<< grid, threads >>> 
		(nbody, ipbuf, jpbuf, fobuf);
#endif

	// recieve force
	// size_t fosize = ni * NJBLOCK * sizeof(Force);
	// hipMemcpy(fo_host, fo_dev, fosize, hipMemcpyDeviceToHost);
	fobuf.dtoh(ni);

	// reduction phase
#pragma omp parallel for
	for(int i=0; i<ni; i++){
		int tid = omp_get_thread_num();
		double ax=0, ay=0, az=0;
		double jx=0, jy=0, jz=0;
#ifdef POTENTIAL
		double poti=0;
#endif
		for(int jb=0; jb<NJBLOCK; jb++){
			// Force &fo = fo_host[i][jb];
			Force &fo = fobuf[i][jb];
			ax += fo.acc.x;
			ay += fo.acc.y;
			az += fo.acc.z;
			jx += fo.jrk.x;
			jy += fo.jrk.y;
			jz += fo.jrk.z;
#ifdef POTENTIAL
			poti += fo.pot;
#endif
		}
		acc[i][0] = ax;
		acc[i][1] = ay;
		acc[i][2] = az;
		jrk[i][0] = jx;
		jrk[i][1] = jy;
		jrk[i][2] = jz;
		// fprintf(stderr, "%f %f %f %f %f %f\n", ax, ay, az, jx, jy, jz);
		// exit(0);
#ifdef POTENTIAL
		pot[i] = poti;
#endif
		bool overflow = false;
		nblist[tid].clear();
		for(int jb=0; jb<NJBLOCK; jb++){
			// Force &fo = fo_host[i][jb];
			Force &fo = fobuf[i][jb];
			int jstart = (nbody * jb) / NJBLOCK;
			if(fo.nnb <= NBMAX){
				for(int k=0; k<fo.nnb; k++){
					int nb = fo.neib[k];
					while(nb < jstart) nb += (1<<16);
					nblist[tid].push_back(nb);
					// nblist.push_back(fo.neib[k]);
				}
			}else{
				overflow = true;
			}
		}
		int *nnbp = listbase + lmax * i;
		int *nblistp = nnbp + 1;
		int nnb = nblist[tid].size();
		if(nnb > nbmax) overflow = true;
		// assert(!overflow);
		if(overflow){
			*nnbp = -1;
		}else{
			*nnbp = nnb;
			for(int k=0; k<nnb; k++){
				nblistp[k] = nblist[tid][k];
			}
		}
	}
#if 0
	if(ni > 0){
		FILE *fp = fopen("Force.gpu", "w");
		assert(fp);
		for(int i=0; i<ni; i++){
			int nnb =  listbase[i*lmax];
			fprintf(fp, "%d %9.2e %9.2e %9.2e %9.2e %9.2e %9.2e %d\n",
					i, acc[i][0], acc[i][1], acc[i][2], 
					   jrk[i][0], jrk[i][1], jrk[i][2], nnb);
		}
		fprintf(fp, "\n");
		fclose(fp);
		exit(1);
	}
#endif
	time_grav += get_wtime();
}

extern "C" {
	void gpunb_open_(int *nbmax){
		GPUNB_open(*nbmax);
	}
	void gpunb_close_(){
		GPUNB_close();
	}
	void gpunb_send_(
			int *nj,
			double mj[],
			double xj[][3],
			double vj[][3]){
		GPUNB_send(*nj, mj, xj, vj);
	}
	void gpunb_regf_(
			int *ni,
			double h2[],
			double xi[][3],
			double vi[][3],
			double acc[][3],
			double jrk[][3],
			double pot[],
			int *lmax,
			int *nbmax,
			int *list){ // list[][lmax]
		GPUNB_regf(*ni, h2, xi, vi, acc, jrk, pot, *lmax, *nbmax, list);
	}
}
