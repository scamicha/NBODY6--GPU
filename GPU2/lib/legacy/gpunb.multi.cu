#include "hip/hip_runtime.h"
// #include <iostream>
#include <cstdio>
// #include <vector>
#include <cmath>
#include <cassert>
#include <cstdlib>
#include <cutil.h>
#include <omp.h>
#include "cuda_pointer.h"

#define NTHREAD 64 // 64, 96, 128 or 192
// #define NJBLOCK 16 // 8800GTS/512 has 16
#define NJBLOCK 14 // for GTX 470
#define NIBLOCK 16 // 16 or 32 
#define NIMAX (NTHREAD * NIBLOCK) // 1024

#define NBMAX 64 // NNB per block, must be power of 2

#define MAX_CPU 8
#define MAX_GPU 4

template <class T>
struct myvector{
	int num;
	T *val;
	myvector(){
		num = 0;
		val = NULL;
	}
	~myvector(){
		delete [] val;
	}
	void clear(){
		num = 0;
	}
	void reserve(size_t count){
		val = new T[count];
	}
	void free(){
		delete [] val;
	}
	void push_back(const T &t){
		val[num++] = t;
	}
	size_t size(){
		return num;
	}
	void resize(size_t size){
		num = size;
	}
	T &operator[](int i){
		return val[i];
	}
};

#define PROFILE
#ifdef PROFILE
#include <sys/time.h>
static double get_wtime(){
#if 1
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.e-6 * tv.tv_usec;
#else
	struct timespec tv;
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &tv);
	return tv.tv_sec + 1.e-9 * tv.tv_nsec;
#endif
}
#else
static double get_wtime(){
	return 0.0;
}
#endif

static double time_send, time_grav, time_reduce;
static long long numInter;

struct Jparticle{
	float3 pos;
	float  mass;
	float3 vel;
	float  pad;
	Jparticle() {}
	Jparticle(double mj, double xj[3], double vj[3]){
		pos.x = xj[0];
		pos.y = xj[1];
		pos.z = xj[2];
		mass  = mj;
		vel.x = vj[0];
		vel.y = vj[1];
		vel.z = vj[2];
	}
};
struct Iparticle{
	float3 pos;
	float  h2;
	float3 vel;
	float  pad;
	Iparticle() {}
	Iparticle(double h2i, double xi[3], double vi[3]){
		pos.x = xi[0];
		pos.y = xi[1];
		pos.z = xi[2];
		h2    = h2i;
		vel.x = vi[0];
		vel.y = vi[1];
		vel.z = vi[2];
	}
};
struct Force{
	float3 acc;
	float  pot;
	float3 jrk;
	int    nnb;          //  8 words
	unsigned short  neib[NBMAX]; // 24 words
	__device__  Force(){
		acc.x = acc.y = acc.z = 0.f;
		jrk.x = jrk.y = jrk.z = 0.f;
		pot = 0.f;
		nnb = 0;
	}
};

__device__ void h4_kernel(
		const int j,
		const Iparticle &ip, 
		const Jparticle &jp, 
		Force &fo,
		float3 &acc,
		float3 &jrk,
		int &nnb){
	float dx = jp.pos.x - ip.pos.x;
	float dy = jp.pos.y - ip.pos.y;
	float dz = jp.pos.z - ip.pos.z;
	float dvx = jp.vel.x - ip.vel.x;
	float dvy = jp.vel.y - ip.vel.y;
	float dvz = jp.vel.z - ip.vel.z;

	float r2 = dx*dx + dy*dy + dz*dz;
	float rv = dx*dvx + dy*dvy + dz*dvz;
	float rinv1 = rsqrtf(r2);
	if(r2 < ip.h2){
		// fo.neib[fo.nnb++ % NBMAX] = j;
		fo.neib[nnb & (NBMAX-1)] = (unsigned)j;
		nnb++;
		rinv1 = 0.f;
	}
	float rinv2 = rinv1 * rinv1;
	float mrinv1 = jp.mass * rinv1;
	float mrinv3 = mrinv1 * rinv2;
	rv *= -3.f * rinv2;
	
#ifdef POTENTIAL
	fo.pot += mrinv1;
#endif
	acc.x += mrinv3 * dx;
	acc.y += mrinv3 * dy;
	acc.z += mrinv3 * dz;
	// fo.acc.z += 1.0;
	jrk.x += mrinv3 * (dvx + rv * dx);
	jrk.y += mrinv3 * (dvy + rv * dy);
	jrk.z += mrinv3 * (dvz + rv * dz);
}

#if 0
__device__ void h4_grav_kernel(
		const int j,
		const Iparticle &ip, 
		const Jparticle &jp, 
		float3 &acc,
		float3 &jrk){
	float dx = jp.pos.x - ip.pos.x;
	float dy = jp.pos.y - ip.pos.y;
	float dz = jp.pos.z - ip.pos.z;
	float dvx = jp.vel.x - ip.vel.x;
	float dvy = jp.vel.y - ip.vel.y;
	float dvz = jp.vel.z - ip.vel.z;

	float r2 = dx*dx + dy*dy + dz*dz;
	float rv = dx*dvx + dy*dvy + dz*dvz;
	float rinv1 = rsqrtf(r2);
	if(r2 < ip.h2){
		rinv1 = 0.f;
	}
	float rinv2 = rinv1 * rinv1;
	float mrinv1 = jp.mass * rinv1;
	float mrinv3 = mrinv1 * rinv2;
	rv *= -3.f * rinv2;
	
#ifdef POTENTIAL
	fo.pot += mrinv1;
#endif
	acc.x += mrinv3 * dx;
	acc.y += mrinv3 * dy;
	acc.z += mrinv3 * dz;
	// fo.acc.z += 1.0;
	jrk.x += mrinv3 * (dvx + rv * dx);
	jrk.y += mrinv3 * (dvy + rv * dy);
	jrk.z += mrinv3 * (dvz + rv * dz);
}

__device__ void h4_neib_kernel(
		const int j,
		const Iparticle &ip, 
		const Jparticle &jp, 
		Force &fo,
		int &nnb){
	float dx = jp.pos.x - ip.pos.x;
	float dy = jp.pos.y - ip.pos.y;
	float dz = jp.pos.z - ip.pos.z;

	float r2 = dx*dx + dy*dy + dz*dz;
	if(r2 < ip.h2){
		// fo.neib[fo.nnb++ % NBMAX] = j;
		fo.neib[nnb & (NBMAX-1)] = (unsigned)j;
		nnb++;
	}
}
#endif

__global__ void h4_gravity(
		int nbody,
		Iparticle ipbuf[],
		Jparticle jpbuf[],
		Force fobuf[][NJBLOCK]){
	int ibid = blockIdx.x;
	int jbid = blockIdx.y;
	int tid = threadIdx.x;
	int iaddr = tid + NTHREAD * ibid;
	int jstart = (nbody * (jbid  )) / NJBLOCK;
	int jend   = (nbody * (jbid+1)) / NJBLOCK;

	Iparticle ip = ipbuf[iaddr];
	// Force fo;
	Force &fo = fobuf[iaddr][jbid];
	float3 acc = make_float3(0.0f, 0.0f, 0.0f);
	float3 jrk = make_float3(0.0f, 0.0f, 0.0f);
	int nnb = 0;
	for(int j=jstart; j<jend; j+=NTHREAD){
		__shared__ Jparticle jpshare[NTHREAD];
		__syncthreads();
#if 0
		jpshare[tid] = jpbuf[j+tid];
#else
		float4 *src = (float4 *)&jpbuf[j];
		float4 *dst = (float4 *)jpshare;
		dst[        tid] = src[        tid];
		dst[NTHREAD+tid] = src[NTHREAD+tid];
#endif
		__syncthreads();

#if 1
		if(jend-j < NTHREAD){
#pragma unroll 4
			for(int jj=0; jj<jend-j; jj++){
				Jparticle jp = jpshare[jj];
				h4_kernel(j+jj, ip, jp, fo, acc, jrk, nnb);
			}
		}else{
#pragma unroll 4
			for(int jj=0; jj<NTHREAD; jj++){
				Jparticle jp = jpshare[jj];
				h4_kernel(j+jj, ip, jp, fo, acc, jrk, nnb);
			}
		}
#else
		if(jend-j < NTHREAD){
#pragma unroll 4
			for(int jj=0; jj<jend-j; jj++){
				Jparticle jp = jpshare[jj];
				h4_grav_kernel(j+jj, ip, jp, acc, jrk);
			}
#pragma unroll 4
			for(int jj=0; jj<jend-j; jj++){
				Jparticle jp = jpshare[jj];
				h4_neib_kernel(j+jj, ip, jp, fo, nnb);
			}
		}else{
#pragma unroll 4
			for(int jj=0; jj<NTHREAD; jj++){
				Jparticle jp = jpshare[jj];
				h4_grav_kernel(j+jj, ip, jp, acc, jrk);
			}
#pragma unroll 4
			for(int jj=0; jj<NTHREAD; jj++){
				Jparticle jp = jpshare[jj];
				h4_neib_kernel(j+jj, ip, jp, fo, nnb);
			}
		}
#endif
	}
	fo.acc = acc;
	fo.jrk = jrk;
	fo.nnb = nnb;
	// fobuf[iaddr][jbid] = fo;
}

static cudaPointer <Jparticle> jpbuf[MAX_GPU];
static cudaPointer <Iparticle> ipbuf[MAX_GPU];
static cudaPointer <Force[NJBLOCK]> fobuf[MAX_GPU];
static int numCPU, numGPU;
static int joff[MAX_GPU + 1];
static myvector<int> nblist[MAX_CPU];
static int nbody, nbodymax;
static int device_id[MAX_GPU];
// static int *nblist;
static bool is_open = false;
static bool devinit = false;

void GPUNB_devinit(){
	if(devinit) return;

	hipGetDeviceCount(&numGPU);
	assert(numGPU <= MAX_GPU);
	char *gpu_list = getenv("GPU_LIST");
	if(gpu_list){
		// get GPU list from environment variable
		numGPU = 0;
		char *p = strtok(gpu_list, " ");
		while(p){
			device_id[numGPU++] = atoi(p);
			p = strtok(NULL, " ");
			assert(numGPU <= MAX_GPU);
		}
	}else{
		// use all GPUs
		for(int i=0; i<numGPU; i++){
			device_id[i] = i;
		}
	}
	
	// numGPU = 1;
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid == 0) numCPU = omp_get_num_threads();
	}
	assert(numCPU <= MAX_CPU);
	assert(numGPU <= numCPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			hipSetDevice(device_id[tid]);
		}
	}
#ifdef PROFILE
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "Initializing NBODY6/GPU library\n");
	fprintf(stderr, "#CPU %d, #GPU %d\n", numCPU, numGPU);
	fprintf(stderr, " device:");
	for(int i=0; i<numGPU; i++){
		fprintf(stderr, " %d", device_id[i]);
	}
	fprintf(stderr, "\n");
	fprintf(stderr, "***********************\n");
#endif
	devinit = true;
}

void GPUNB_open(int nbmax){
	time_send = time_grav = time_reduce = 0.0;
	numInter = 0;
	nbodymax = nbmax;

	if(is_open){
		fprintf(stderr, "gpunb: it is already open\n");
		return;
	}
	is_open = true;

#if 0
	hipGetDeviceCount(&numGPU);
	assert(numGPU <= MAX_GPU);
	char *gpu_list = getenv("GPU_LIST");
	if(gpu_list){
		// get GPU list from environment variable
		numGPU = 0;
		char *p = strtok(gpu_list, " ");
		while(p){
			device_id[numGPU++] = atoi(p);
			p = strtok(NULL, " ");
			assert(numGPU <= MAX_GPU);
		}
	}else{
		// use all GPUs
		for(int i=0; i<numGPU; i++){
			device_id[i] = i;
		}
	}
	
	// numGPU = 1;
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid == 0) numCPU = omp_get_num_threads();
	}
	assert(numCPU <= MAX_CPU);
	assert(numGPU <= numCPU);
#else
	GPUNB_devinit();
#endif

	for(int id=0; id<numGPU + 1; id++){
		joff[id] = (id * nbmax) / numGPU;
	}

	// omp_set_num_threads(numGPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			hipSetDevice(device_id[tid]);
			int nj = joff[tid+1] - joff[tid];
			jpbuf[tid].allocate(nj + NTHREAD);
			ipbuf[tid].allocate(NIMAX);
			fobuf[tid].allocate(NIMAX);
		}
	}
	// omp_set_num_threads(numCPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		nblist[tid].reserve(nbmax);
	}
#ifdef PROFILE
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "Opened NBODY6/GPU library\n");
	fprintf(stderr, "#CPU %d, #GPU %d\n", numCPU, numGPU);
	fprintf(stderr, " device:");
	for(int i=0; i<numGPU; i++){
		fprintf(stderr, " %d", device_id[i]);
	}
	fprintf(stderr, "\n");
	for(int i=0; i<numGPU+1; i++){
		fprintf(stderr, " %d", joff[i]);
	}
	fprintf(stderr, "\n");
	fprintf(stderr, "nbmax = %d\n", nbmax);
	fprintf(stderr, "***********************\n");
#endif
}

void GPUNB_close(){
	if(!is_open){
		fprintf(stderr, "gpunb: it is already close\n");
		return;
	}
	is_open = false;
	// omp_set_num_threads(numGPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			jpbuf[tid].free();
			ipbuf[tid].free();
			fobuf[tid].free();
		}
	}
	// omp_set_num_threads(numCPU);
	nbodymax = 0;

#ifdef PROFILE
# if 0
	std::cerr << "***********************" << std::endl;
	std::cerr << "time send : " << time_send << " sec " << std::endl;
	std::cerr << "time grav : " << time_grav << " sec " << std::endl;
	std::cerr << 60.e-9 * numInter / time_grav << " Gflops (gravity part only)" << std::endl;
	std::cerr << "***********************" << std::endl;
# else
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "time send   : %f sec\n", time_send);
	fprintf(stderr, "time grav   : %f sec\n", time_grav);
	fprintf(stderr, "time reduce : %f sec\n", time_reduce);
	fprintf(stderr, "%f Gflops (gravity part only)\n", 60.e-9 * numInter / time_grav);
	fprintf(stderr, "***********************\n");
# endif
#endif
}

void GPUNB_send(
		int _nbody,
		double mj[],
		double xj[][3],
		double vj[][3]){
	nbody = _nbody;
	assert(nbody <= nbodymax);
	time_send -= get_wtime();
	for(int id=0; id<numGPU + 1; id++){
		joff[id] = (id * nbody) / numGPU;
	}
	// omp_set_num_threads(numGPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			int nj = joff[tid+1] - joff[tid];
			// fprintf(stderr, "%d : %d\n", tid, nj);
			for(int j=0; j<nj; j++){
				int jj = j + joff[tid];
				jpbuf[tid][j] = Jparticle(mj[jj], xj[jj], vj[jj]);
			}
			jpbuf[tid].htod(nj);
		}
	}
	// size_t jpsize = nj * sizeof(Jparticle);
	// hipMemcpy(jp_dev, jp_host, jpsize, hipMemcpyHostToDevice);
	time_send += get_wtime();
	// omp_set_num_threads(numCPU);
}

static void handle_overflow(
		myvector<int> &list,
		cudaPointer <Iparticle> &ip,
		cudaPointer <Jparticle> &jp,
		int nj,
		int i,
		int jb,
		int nnb,
		int joff
		){
	fprintf(stderr, "gpunb overflow: %d %d %d\n", i, jb, nnb);
	int jstart = (nj * (jb  )) / NJBLOCK;
	int jend   = (nj * (jb+1)) / NJBLOCK;

	float xi = ip[i].pos.x;
	float yi = ip[i].pos.y;
	float zi = ip[i].pos.z;
	float h2i = ip[i].h2;
	int n = 0;
	for(int j=jstart; j<jend; j++){
		float dx = jp[j].pos.x - xi;
		float dy = jp[j].pos.y - yi;
		float dz = jp[j].pos.z - zi;
		float r2 = dx*dx + dy*dy + dz*dz;
		if(r2 < h2i){
			list.push_back(j + joff);
			n++;
		}
	}
	// assert(n == nnb);
	if(n != nnb){
		fprintf(stderr, "warning, NNB_GPU != NNB_CPU\n");
		// list.resize(list.size() - n);
		list.num -= n;
		if(n > nnb){
			ip[i].h2 *= .99999988079071044922f;
		}
		if(n < nnb){
			ip[i].h2 *= 1.00000011920928955078f;
		}
		handle_overflow(list, ip, jp, nj, i, jb, nnb, joff);
	}
}

void GPUNB_regf(
		int ni,
		double h2[],
		double xi[][3],
		double vi[][3],
		double acc[][3],
		double jrk[][3],
		double pot[],
		int lmax,
		int nbmax,
		int *listbase){
	time_grav -= get_wtime();
	numInter += ni * nbody;
	assert(0 < ni && ni <= NIMAX);

	// omp_set_num_threads(numGPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			// hipSetDevice(device_id[tid]);
			int dev;
			hipGetDevice(&dev);
			assert(dev == device_id[tid]);
			for(int i=0; i<ni; i++){
				ipbuf[tid][i] = Iparticle(h2[i], xi[i], vi[i]);
			}
			// set i-particles
			ipbuf[tid].htod(ni);

			// gravity kernel
			int niblock = 1 + (ni-1) / NTHREAD;
			dim3 grid(niblock, NJBLOCK, 1);
			dim3 threads(NTHREAD, 1, 1);
			int nj = joff[tid+1] - joff[tid];
			h4_gravity <<< grid, threads >>> 
				(nj, ipbuf[tid], jpbuf[tid], fobuf[tid]);

			// recieve force
			// fprintf(stderr, "DBG %d %d\n", tid, device_id[tid]);
			fobuf[tid].dtoh(ni);
		}
	}

	const double wt = get_wtime();
	time_grav   += wt;
	time_reduce -= wt;

	// reduction phase
	// omp_set_num_threads(numCPU);
#pragma omp parallel for
	for(int i=0; i<ni; i++){
		int tid = omp_get_thread_num();
		double ax=0, ay=0, az=0;
		double jx=0, jy=0, jz=0;
#ifdef POTENTIAL
		double poti=0;
#endif
		for(int id=0; id<numGPU; id++){
			for(int jb=0; jb<NJBLOCK; jb++){
				// Force &fo = fo_host[i][jb];
				Force &fo = fobuf[id][i][jb];
				ax += fo.acc.x;
				ay += fo.acc.y;
				az += fo.acc.z;
				jx += fo.jrk.x;
				jy += fo.jrk.y;
				jz += fo.jrk.z;
#ifdef POTENTIAL
				poti += fo.pot;
#endif
			}
		}
		acc[i][0] = ax;
		acc[i][1] = ay;
		acc[i][2] = az;
		jrk[i][0] = jx;
		jrk[i][1] = jy;
		jrk[i][2] = jz;
		// fprintf(stderr, "%f %f %f %f %f %f\n", ax, ay, az, jx, jy, jz);
		// exit(0);
#ifdef POTENTIAL
		pot[i] = poti;
#endif
		bool overflow = false;
		nblist[tid].clear();
		for(int id=0; id<numGPU; id++){
			for(int jb=0; jb<NJBLOCK; jb++){
				// Force &fo = fo_host[i][jb];
				Force &fo = fobuf[id][i][jb];
				int nj = joff[id+1] - joff[id];
				// int jstart = (nbody * jb) / NJBLOCK;
				int jstart = (nj * jb) / NJBLOCK;
				if(fo.nnb <= NBMAX){
					for(int k=0; k<fo.nnb; k++){
						int nb = fo.neib[k];
						while(nb < jstart) nb += (1<<16);
						nb += joff[id];
						nblist[tid].push_back(nb);
						// nblist.push_back(fo.neib[k]);
					}
				}else{
					// overflow = true;
					handle_overflow(nblist[tid], ipbuf[id], jpbuf[id], nj, i, jb, fo.nnb, joff[id]);
				}
			}
		}
		int *nnbp = listbase + lmax * i;
		int *nblistp = nnbp + 1;
		int nnb = nblist[tid].size();
		if(nnb > nbmax) overflow = true;
		// assert(!overflow);
		if(overflow){
			*nnbp = -1;
		}else{
			*nnbp = nnb;
			for(int k=0; k<nnb; k++){
				nblistp[k] = nblist[tid][k];
			}
		}
	}
#if 0
	if(ni > 0){
		FILE *fp = fopen("Force.gpu", "w");
		assert(fp);
		for(int i=0; i<ni; i++){
			int nnb =  listbase[i*lmax];
			fprintf(fp, "%d %9.2e %9.2e %9.2e %9.2e %9.2e %9.2e %d\n",
					i, acc[i][0], acc[i][1], acc[i][2], 
					   jrk[i][0], jrk[i][1], jrk[i][2], nnb);
		}
		fprintf(fp, "\n");
		fclose(fp);
		exit(1);
	}
#endif
	// time_grav += get_wtime();
	time_reduce += get_wtime();
}

extern "C" {
	void gpunb_devinit_(){
		GPUNB_devinit();
	}
	void gpunb_open_(int *nbmax){
		GPUNB_open(*nbmax);
	}
	void gpunb_close_(){
		GPUNB_close();
	}
	void gpunb_send_(
			int *nj,
			double mj[],
			double xj[][3],
			double vj[][3]){
		GPUNB_send(*nj, mj, xj, vj);
	}
	void gpunb_regf_(
			int *ni,
			double h2[],
			double xi[][3],
			double vi[][3],
			double acc[][3],
			double jrk[][3],
			double pot[],
			int *lmax,
			int *nbmax,
			int *list){ // list[][lmax]
		GPUNB_regf(*ni, h2, xi, vi, acc, jrk, pot, *lmax, *nbmax, list);
	}
}
