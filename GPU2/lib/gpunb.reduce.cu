#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <cassert>
#include <cstdlib>
#include <cutil.h>
#include <omp.h>
#include "cuda_pointer.h"

#define NTHREAD 64 // 64 or 128
#define NJBLOCK 14 // for GTX 470
#define NIBLOCK 32 // 16 or 32 
#define NIMAX (NTHREAD * NIBLOCK) // 2048

#define NXREDUCE 16 // must be >NJBLOCK
#define NYREDUCE  8

#define NNB_PER_BLOCK 256 // NNB per block, must be power of 2
#define NB_BUF_SIZE (1<<20)

#define MAX_CPU 8
#define MAX_GPU 3

// for clarity, for myself
#define __out

#define PROFILE

#define NAN_CHECK(val) assert((val) == (val));

typedef unsigned short uint16;

struct Jparticle{
	float3 pos;
	float  mass;
	float3 vel;
	float  pad;
	Jparticle() {}
	Jparticle(double mj, double xj[3], double vj[3]){
		pos.x = xj[0];
		pos.y = xj[1];
		pos.z = xj[2];
		mass  = mj;
		vel.x = vj[0];
		vel.y = vj[1];
		vel.z = vj[2];

		NAN_CHECK(xj[0]);
		NAN_CHECK(xj[1]);
		NAN_CHECK(xj[2]);
		NAN_CHECK(mj);
		NAN_CHECK(vj[0]);
		NAN_CHECK(vj[1]);
		NAN_CHECK(vj[2]);
	}
};
struct Iparticle{
	float3 pos;
	float  h2;
	float3 vel;
	float  pad;
	Iparticle() {}
	Iparticle(double h2i, double xi[3], double vi[3]){
		pos.x = xi[0];
		pos.y = xi[1];
		pos.z = xi[2];
		h2    = h2i;
		vel.x = vi[0];
		vel.y = vi[1];
		vel.z = vi[2];

		NAN_CHECK(xi[0]);
		NAN_CHECK(xi[1]);
		NAN_CHECK(xi[2]);
		NAN_CHECK(h2i);
		NAN_CHECK(vi[0]);
		NAN_CHECK(vi[1]);
		NAN_CHECK(vi[2]);
	}
};
struct Force{
	float3 acc;
	float  pot;
	float3 jrk;
	int    nnb;          //  8 words
	__device__  void clear(){
		acc.x = acc.y = acc.z = 0.f;
		jrk.x = jrk.y = jrk.z = 0.f;
		pot = 0.f;
		nnb = 0;
	}
	__device__ void operator+=(const Force &rhs){
		acc.x += rhs.acc.x;
		acc.y += rhs.acc.y;
		acc.z += rhs.acc.z;
		pot   += rhs.pot;
		jrk.x += rhs.jrk.x;
		jrk.y += rhs.jrk.y;
		jrk.z += rhs.jrk.z;
		if(nnb>=0 && rhs.nnb>=0){
			nnb += rhs.nnb;
		}else{
			nnb = -1;
		}
	}
};

__device__ void dev_gravity(
		const int        jidx,
		const Iparticle &ip, 
		const Jparticle &jp, 
		__out Force     &fo,
		__out uint16     nblist[]){
	float dx = jp.pos.x - ip.pos.x;
	float dy = jp.pos.y - ip.pos.y;
	float dz = jp.pos.z - ip.pos.z;
	float dvx = jp.vel.x - ip.vel.x;
	float dvy = jp.vel.y - ip.vel.y;
	float dvz = jp.vel.z - ip.vel.z;

	float r2 = dx*dx + dy*dy + dz*dz;
	float rv = dx*dvx + dy*dvy + dz*dvz;
	float rinv1 = rsqrtf(r2);
	if(r2 < ip.h2){
		// fo.neib[fo.nnb++ % NBMAX] = j;
		nblist[fo.nnb & (NNB_PER_BLOCK-1)] = (uint16)jidx;
		fo.nnb++;
		rinv1 = 0.f;
	}
	float rinv2 = rinv1 * rinv1;
	float mrinv1 = jp.mass * rinv1;
	float mrinv3 = mrinv1 * rinv2;
	rv *= -3.f * rinv2;
	
#ifdef POTENTIAL
	fo.pot += mrinv1;
#endif
	fo.acc.x += mrinv3 * dx;
	fo.acc.y += mrinv3 * dy;
	fo.acc.z += mrinv3 * dz;
	// fo.acc.z += 1.0;
	fo.jrk.x += mrinv3 * (dvx + rv * dx);
	fo.jrk.y += mrinv3 * (dvy + rv * dy);
	fo.jrk.z += mrinv3 * (dvz + rv * dz);
}

__global__ void gravity_kernel(
		const int       nbody,
		const Iparticle ipbuf[],
		const Jparticle jpbuf[],
		__out Force     fobuf[][NJBLOCK],
		__out uint16    nbbuf[][NJBLOCK][NNB_PER_BLOCK]){
	int ibid = blockIdx.x;
	int jbid = blockIdx.y;
	int tid = threadIdx.x;
	int iaddr = tid + blockDim.x * ibid;
	int jstart = (nbody * (jbid  )) / NJBLOCK;
	int jend   = (nbody * (jbid+1)) / NJBLOCK;

	Iparticle ip = ipbuf[iaddr];
	Force fo;
	fo.clear();
	uint16 *nblist = nbbuf[iaddr][jbid];
	for(int j=jstart; j<jend; j+=NTHREAD){
		__shared__ Jparticle jpshare[NTHREAD];
		__syncthreads();
#if 0
		jpshare[tid] = jpbuf[j+tid];
#else
		float4 *src = (float4 *)&jpbuf[j];
		float4 *dst = (float4 *)jpshare;
		dst[        tid] = src[        tid];
		dst[NTHREAD+tid] = src[NTHREAD+tid];
#endif
		__syncthreads();

		if(jend-j < NTHREAD){
#pragma unroll 4
			for(int jj=0; jj<jend-j; jj++){
				Jparticle jp = jpshare[jj];
				dev_gravity(j-jstart+jj, ip, jp, fo, nblist);
			}
		}else{
#pragma unroll 4
			for(int jj=0; jj<NTHREAD; jj++){
				Jparticle jp = jpshare[jj];
				dev_gravity(j-jstart+jj, ip, jp, fo, nblist);
			}
		}
	}
	if(fo.nnb > NNB_PER_BLOCK) fo.nnb = -1;
	fobuf[iaddr][jbid] = fo;
}

#if 0
__global__ void reduce_kernel_old(
		const int     nbody,
		const int     joff,
		// here's partial forces and nblists,
		const Force   fpart [][NJBLOCK],
		const uint16  nbpart[][NJBLOCK][NNB_PER_BLOCK],
		// and these to be redeced
		Force         ftot    [],
		int           nbtot   [][NNB_MAX]){
	const int ibid = blockIdx.x;
	int tid = threadIdx.x;
	const int iaddr = tid + blockDim.x * ibid;

	Force fo;
	fo.clear();
	int *nbdst   = nbtot[iaddr];
	bool oveflow = false;

	for(int jb=0; jb<NJBLOCK; jb++){
		const int jstart = (nbody * jb) / NJBLOCK;
		const Force &fsrc = fpart[iaddr][jb];
		fo += fsrc;
		if(fsrc.nnb > NNB_PER_BLOCK) oveflow = true;
		if(fo.nnb   > NNB_MAX      ) oveflow = true;
		if(!oveflow){
			const int klen = fsrc.nnb;
			for(int k=0; k<klen; k++){
				const int nbid = (joff + jstart) + int(nbpart[iaddr][jb][k]);
				*nbdst++ = nbid;
			}
		}
	}
	if(oveflow) fo.nnb = -1;
	ftot[iaddr] = fo;
}
#endif

__global__ void force_reduce_kernel(
		const int ni,
		const Force fpart[][NJBLOCK],
		__out Force ftot []){
	const int xid = threadIdx.x;
	const int yid = threadIdx.y;
	const int bid = blockIdx.x;
	const int iaddr = yid + blockDim.y * bid;

	__shared__ Force fshare[NYREDUCE][NXREDUCE];
	if(xid < NJBLOCK){
		fshare[yid][xid] = fpart[iaddr][xid];
	}else{
		fshare[yid][xid].clear();
	}
	Force *fs = fshare[yid];
#if NXREDUCE==32
	if(xid < 16) fs[xid] += fs[xid + 16];
#endif
	if(xid < 8) fs[xid] += fs[xid + 8];
	if(xid < 4) fs[xid] += fs[xid + 4];
	if(xid < 2) fs[xid] += fs[xid + 2];
	if(xid < 1) fs[xid] += fs[xid + 1];
	
	if(iaddr < ni){
		ftot[iaddr] = fs[0];
	}
}

__global__ void gather_nb_kernel(
		const int    ni,
		const int    nj,
		const int    joff,
		const Force  fpart[][NJBLOCK],
		const Force  ftot [],
		const int    nboff[],
		const uint16 nbpart[][NJBLOCK][NNB_PER_BLOCK],
		__out   int  nblist[])
{
	const int xid = threadIdx.x;
	const int yid = threadIdx.y;
	const int bid = blockIdx.x;
	const int iaddr = yid + blockDim.y * bid;
	if(iaddr >= ni) return;
	if(ftot[iaddr].nnb < 0) return;

	const int mynnb = (xid < NJBLOCK) ? fpart[iaddr][xid].nnb
	                                  : 0;

	// now performe prefix sum
	__shared__ int ishare[NYREDUCE][NXREDUCE];
	ishare[yid][xid] = mynnb;
	int *ish = ishare[yid];
	if(xid>=1)  ish[xid] += ish[xid-1];
	if(xid>=2)  ish[xid] += ish[xid-2];
	if(xid>=4)  ish[xid] += ish[xid-4];
	if(xid>=8)  ish[xid] += ish[xid-8];
#if NXREDUCE==32
	if(xid>=16)  ish[xid] += ish[xid-16];
#endif

	const int off = (xid == 0) ? 0 
	                           : ish[xid-1];
	int *nbdst = nblist + nboff[iaddr] + off;

	const int jstart = (nj * xid) / NJBLOCK;
	if(xid < NJBLOCK){
		for(int k=0; k<mynnb; k++){
			const int nbid = (joff + jstart) + int(nbpart[iaddr][xid][k]);
			// const int nbid = iaddr * 1000 + k;
			nbdst[k] = nbid;
		}
	}
}


/*// Host Part
#ifdef PROFILE
#include <sys/time.h>
static double get_wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.e-6 * tv.tv_usec;
}
#else
static double get_wtime(){
	return 0.0;
}
#endif

static double time_send, time_grav, time_reduce;
static long long numInter;
static cudaPointer <Jparticle> jpbuf[MAX_GPU];
static cudaPointer <Iparticle> ipbuf[MAX_GPU];
static cudaPointer <Force[NJBLOCK]> fpart[MAX_GPU];
static cudaPointer <Force>          ftot [MAX_GPU];
static cudaPointer <uint16[NJBLOCK][NNB_PER_BLOCK]> nbpart[MAX_GPU];
static cudaPointer <int> nblist [MAX_GPU];
static cudaPointer <int> nboff  [MAX_GPU];
static int numCPU, numGPU;
static int joff[MAX_GPU + 1];
static int nbody, nbodymax;
static int devid[MAX_GPU];
static bool is_open = false;
static bool devinit = false;

void GPUNB_devinit(){
	if(devinit) return;

	assert(NXREDUCE >= NJBLOCK);
	assert(NXREDUCE <= 32);

	hipGetDeviceCount(&numGPU);
	assert(numGPU <= MAX_GPU);
	char *gpu_list = getenv("GPU_LIST");
	if(gpu_list){
		// get GPU list from environment variable
		numGPU = 0;
		char *p = strtok(gpu_list, " ");
		while(p){
			devid[numGPU++] = atoi(p);
			p = strtok(NULL, " ");
			assert(numGPU <= MAX_GPU);
		}
	}else{
		// use all GPUs
		for(int i=0; i<numGPU; i++){
			devid[i] = i;
		}
	}
	
	// numGPU = 1;
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid == 0) numCPU = omp_get_num_threads();
	}
	assert(numCPU <= MAX_CPU);
	assert(numGPU <= numCPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			hipSetDevice(devid[tid]);
		}
	}
#ifdef PROFILE
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "Initializing NBODY6/GPU library\n");
	fprintf(stderr, "#CPU %d, #GPU %d\n", numCPU, numGPU);
	fprintf(stderr, " device:");
	for(int i=0; i<numGPU; i++){
		fprintf(stderr, " %d", devid[i]);
	}
	fprintf(stderr, "\n");
#if 1
	for(int i=0; i<numGPU; i++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, devid[i]);
		fprintf(stderr, " device %d: %s\n", devid[i], prop.name);
	}
#endif
	fprintf(stderr, "***********************\n");
#endif
	devinit = true;
}

void GPUNB_open(int nbmax){
	time_send = time_grav = time_reduce = 0.0;
	numInter = 0;
	nbodymax = nbmax;

	GPUNB_devinit();

	if(is_open){
		fprintf(stderr, "gpunb: it is already open\n");
		return;
	}
	is_open = true;


	for(int id=0; id<numGPU + 1; id++){
		joff[id] = (id * nbmax) / numGPU;
	}

	// omp_set_num_threads(numGPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			hipSetDevice(devid[tid]);
			int nj = joff[tid+1] - joff[tid];
			jpbuf [tid].allocate(nj + NTHREAD);
			ipbuf [tid].allocate(NIMAX);
			fpart [tid].allocate(NIMAX);
			ftot  [tid].allocate(NIMAX);
			nbpart[tid].allocate(NIMAX);
			nblist[tid].allocate(NB_BUF_SIZE); // total ganged nblist
			nboff [tid].allocate(NIMAX+1);
		}
	}
#ifdef PROFILE
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "Opened NBODY6/GPU library\n");
	fprintf(stderr, "#CPU %d, #GPU %d\n", numCPU, numGPU);
	fprintf(stderr, " device:");
	for(int i=0; i<numGPU; i++){
		fprintf(stderr, " %d", devid[i]);
	}
	fprintf(stderr, "\n");
	for(int i=0; i<numGPU+1; i++){
		fprintf(stderr, " %d", joff[i]);
	}
	fprintf(stderr, "\n");
	fprintf(stderr, "nbmax = %d\n", nbmax);
	fprintf(stderr, "***********************\n");
#endif
}

void GPUNB_close(){
	if(!is_open){
		fprintf(stderr, "gpunb: it is already close\n");
		return;
	}
	is_open = false;
	// omp_set_num_threads(numGPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			jpbuf [tid].free();
			ipbuf [tid].free();
			fpart [tid].free();
			ftot  [tid].free();
			nbpart[tid].free();
			nblist[tid].free();
			nboff [tid].free();
		}
	}
	// omp_set_num_threads(numCPU);
	nbodymax = 0;

#ifdef PROFILE
	fprintf(stderr, "Closed NBODY6/GPU library\n");
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "time send   : %f sec\n", time_send);
	fprintf(stderr, "time grav   : %f sec\n", time_grav);
	fprintf(stderr, "time reduce : %f sec\n", time_reduce);
	fprintf(stderr, "%f Gflops (gravity part only)\n", 60.e-9 * numInter / time_grav);
	fprintf(stderr, "***********************\n");
#endif
}

void GPUNB_send(
		int _nbody,
		double mj[],
		double xj[][3],
		double vj[][3]){
	assert(is_open);
	nbody = _nbody;
	assert(nbody <= nbodymax);
	time_send -= get_wtime();
	for(int id=0; id<numGPU + 1; id++){
		joff[id] = (id * nbody) / numGPU;
	}
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			int nj = joff[tid+1] - joff[tid];
			for(int j=0; j<nj; j++){
				int jj = j + joff[tid];
				jpbuf[tid][j] = Jparticle(mj[jj], xj[jj], vj[jj]);
			}
			jpbuf[tid].htod(nj);
		}
	}
	time_send += get_wtime();
}

void GPUNB_regf(
		int ni,
		double h2[],
		double xi[][3],
		double vi[][3],
		double acc[][3],
		double jrk[][3],
		double pot[],
		int lmax,
		int nbmax,
		int *listbase){
	assert(is_open);

	time_grav -= get_wtime();
	numInter += ni * nbody;
	assert(0 < ni && ni <= NIMAX);

	// omp_set_num_threads(numGPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			// hipSetDevice(device_id[tid]);
			for(int i=0; i<ni; i++){
				ipbuf[tid][i] = Iparticle(h2[i], xi[i], vi[i]);
			}
			// set i-particles
			ipbuf[tid].htod(ni);

			// gravity kernel
			int niblock = 1 + (ni-1) / NTHREAD;
			dim3 grid(niblock, NJBLOCK, 1);
			dim3 threads(NTHREAD, 1, 1);
			int nj = joff[tid+1] - joff[tid];
			gravity_kernel <<< grid, threads >>> 
				(nj, ipbuf[tid], jpbuf[tid], fpart[tid], nbpart[tid]);
			// CUDA_SAFE_THREAD_SYNC();

#if 0
			dim3 rgrid(niblock, 1, 1);
			reduce_kernel <<< rgrid, threads >>>
				(nj, joff[tid], fpart[tid], nbpart[tid], ftot[tid], nbtot[tid]);
#else
			const int ni8 = 1 + (ni-1) / NYREDUCE;
			dim3 rgrid   (ni8, 1, 1);
			dim3 rthreads(NXREDUCE, NYREDUCE, 1);
			force_reduce_kernel <<< rgrid, rthreads >>>
				(ni, fpart[tid], ftot[tid]);
#endif
			// CUDA_SAFE_THREAD_SYNC();
			ftot [tid].dtoh(ni);

			// now make prefix sum
			int nbsum = 0;
			for(int i=0; i<ni; i++){
				nboff[tid][i] = nbsum;
				const int nnb = ftot[tid][i].nnb;
				// assert(nnb >= 0);
				if(nnb >= 0) nbsum += nnb;
			}
			assert(nbsum <= NB_BUF_SIZE);
			nboff[tid].htod(ni);

			// debugging
			// for(int k=0; k<nbsum; k++) nblist[tid][k] = -1;
			// nblist[tid].htod(nbsum);

			gather_nb_kernel <<< rgrid, rthreads>>>
				(ni, nj, joff[tid], fpart[tid], ftot[tid], 
				 nboff[tid], nbpart[tid], nblist[tid]);
			// CUDA_SAFE_THREAD_SYNC();
			nblist[tid].dtoh(nbsum);
		}
	}

	const double wt = get_wtime();
	time_grav   += wt;
	time_reduce -= wt;

	// reduction phase
	// omp_set_num_threads(numCPU);
#pragma omp parallel for
	for(int i=0; i<ni; i++){
		double ax=0.0, ay=0.0, az=0.0;
		double jx=0.0, jy=0.0, jz=0.0;
		double po=0.0;

		for(int id=0; id<numGPU; id++){
			Force &fo = ftot[id][i];
			ax += fo.acc.x;
			ay += fo.acc.y;
			az += fo.acc.z;
			jx += fo.jrk.x;
			jy += fo.jrk.y;
			jz += fo.jrk.z;
			po += fo.pot;
		}
		acc[i][0] = ax;
		acc[i][1] = ay;
		acc[i][2] = az;
		jrk[i][0] = jx;
		jrk[i][1] = jy;
		jrk[i][2] = jz;
		pot[i]    = po;
	}
#pragma omp parallel for
	for(int i=0; i<ni; i++){
		bool overflow = false;
		int *nnbp = listbase + lmax * i;
		int *nblistp = nnbp + 1;
		int nnb = 0;
		for(int id=0; id<numGPU; id++){
			const int nnb_part = ftot[id][i].nnb;
			if(nnb_part < 0) overflow = true;
			// assert(!overflow);
			nnb += nnb_part;
			if(nnb > nbmax) overflow = true;
			// assert(!overflow);
			if(!overflow){
				const int off = nboff[id][i]; 
				for(int k=0; k<nnb_part; k++){
					*nblistp++ = nblist[id][off + k];
				}
			}
		}
		if(overflow){
			*nnbp = -1;
		}else{
			*nnbp = nnb;
		}
	}
	time_reduce += get_wtime();
}

extern "C" {
	void gpunb_devinit_(){
		GPUNB_devinit();
	}
	void gpunb_open_(int *nbmax){
		GPUNB_open(*nbmax);
	}
	void gpunb_close_(){
		GPUNB_close();
	}
	void gpunb_send_(
			int *nj,
			double mj[],
			double xj[][3],
			double vj[][3]){
		GPUNB_send(*nj, mj, xj, vj);
	}
	void gpunb_regf_(
			int *ni,
			double h2[],
			double xi[][3],
			double vi[][3],
			double acc[][3],
			double jrk[][3],
			double pot[],
			int *lmax,
			int *nbmax,
			int *list){ // list[][lmax]
		GPUNB_regf(*ni, h2, xi, vi, acc, jrk, pot, *lmax, *nbmax, list);
	}
}*/

